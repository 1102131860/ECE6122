#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>

#define DEFAULT_NUM_THREAD_PER_BLOCK 32
#define DEFAULT_CELL_SIZE 5
#define DEFAUlT_WINDOW_WEIGHT 800
#define DEFAULT_WINDOW_HEIGHT 600
#define RAND_SEED 42

enum class MemoryType {
	NORMAL = 0,
	PINNED = 1,
	MANAGED = 2
};

int numThreadPerBlock = DEFAULT_NUM_THREAD_PER_BLOCK;
int cellSize = DEFAULT_CELL_SIZE;
int windowWeight = DEFAUlT_WINDOW_WEIGHT;
int windowHeight = DEFAULT_WINDOW_HEIGHT;
MemoryType memoryType = MemoryType::NORMAL;

int columnLength = windowWeight / cellSize;
int rowLength = windowHeight / cellSize;
int totalGrid = columnLength * rowLength;
int numBlocks = (totalGrid + numThreadPerBlock - 1) / numThreadPerBlock;

std::vector<int> hostGrid;
int* d_grid_current, *d_grid_next;

// Error checking macro for CUDA calls
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(code) << " in " << file << " at line " << line << std::endl;
        if (abort) exit(code);
    }
}

// Update 1-D Grid
__global__ void updateSingleGridKernelWithHalo(int* grid, int* newGrid, int columnLength, int rowLength) {
    // Calculate the global 1D index for the current thread
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Map the 1D index to 2D coordinates (globalX and globalY)
    int globalX = globalIndex % columnLength; // X-coordinate
    int globalY = globalIndex / columnLength; // Y-coordinate

    // Check if the current thread is within the grid boundaries
    if (globalX < columnLength && globalY < rowLength) {
        // Calculate the number of live neighbors
        int numOfLiveNeighbours = 0;

        // Iterate over all neighbors (including diagonals)
        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                // Skip the center cell itself
                if (dx == 0 && dy == 0) continue;

                int neighborX = globalX + dx;
                int neighborY = globalY + dy;

                // Check if the neighbor is within the grid boundaries
                if (neighborX >= 0 && neighborX < columnLength && neighborY >= 0 && neighborY < rowLength) {
                    numOfLiveNeighbours += grid[neighborY * columnLength + neighborX];
                }
            }
        }

        // Apply the rules of the game and write to the new grid
        if (grid[globalY * columnLength + globalX] == 1) {
            newGrid[globalY * columnLength + globalX] = (numOfLiveNeighbours == 2 || numOfLiveNeighbours == 3) ? 1 : 0;
        } else {
            newGrid[globalY * columnLength + globalX] = (numOfLiveNeighbours == 3) ? 1 : 0;
        }
    }
}

// Obtain arguments through inputs
void argumentsParse(int argc, char* argv[]) {
    for (int i = 1; i < argc; i += 2) {
        std::string argKey = argv[i];
        std::string argValue = (i + 1 < argc) ? argv[i + 1] : "";

        if (argKey == "-n" && !argValue.empty()) {
            try {
                numThreadPerBlock = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-n): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
                std::cout << "Out of range (-n): " << e.what() << std::endl;
            }
            if (numThreadPerBlock <= 0 || numThreadPerBlock % 32 != 0) {
                std::cout << "The number of threads per block must be a multiple of 32" << std::endl;
                numThreadPerBlock = DEFAULT_NUM_THREAD_PER_BLOCK;
            }
        } else if (argKey == "-c" && !argValue.empty()) {
            try {
	            cellSize = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-c): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
	            std::cout << "Out of range (-c): " << e.what() << std::endl;
            }
            if (cellSize < 1) {
                std::cout << "Cell size must be larger than or equal to 1" << std::endl;
                cellSize = DEFAULT_CELL_SIZE;
            }
        } else if (argKey == "-x" && !argValue.empty()) {
            try {
                windowWeight = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-x): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
                std::cout << "Out of range (-x): " << e.what() << std::endl;
            } if (windowWeight < cellSize) {
                std::cout << "Winodw weight mush be larger than or equal to cell size" << std::endl;
                windowWeight = DEFAUlT_WINDOW_WEIGHT;
            }
        } else if (argKey == "-y" && !argValue.empty()) {
            try {
                windowHeight = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-y): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
                std::cout << "Out of range (-y): " << e.what() << std::endl;
            }
            if (windowHeight < cellSize) {
                std::cout << "Winodw height mush be larger than or equal to cell size" << std::endl;
                windowHeight = DEFAULT_WINDOW_HEIGHT;
            }
        } else if (argKey == "-t" && !argValue.empty()) {
            if (argValue == "NORMAL"){
                memoryType = MemoryType::NORMAL;
            } else if (argValue == "PINNED") {
                memoryType = MemoryType::PINNED;
            } else if (argValue == "MANAGED") {
                memoryType = MemoryType::MANAGED;
            } else {
                std::cout << "Invalid argument (-t): select a type from NORMAL, PINNED and MANAGED" << std::endl;
                memoryType = MemoryType::NORMAL;
            }
        }
    }
}

// Draw the gird using RectangleShape
void drawGrid(sf::RenderWindow& window, int* grid) {
    window.clear(sf::Color::Black);
    for (int x = 0; x < columnLength; ++x) {
        for (int y = 0; y < rowLength; ++y) {
            if (grid[y * columnLength + x] == 1) {
                sf::RectangleShape cell(sf::Vector2f(cellSize, cellSize));
                cell.setPosition(x * cellSize, y * cellSize);
                cell.setFillColor(sf::Color::White);
                window.draw(cell);
            }
        }
    }
    window.display();
}

// Normal memory transfer
void normalMemory(sf::RenderWindow& window) {
    // Create CUDA streams
    hipStream_t stream1, stream2;
    cudaCheckError(hipStreamCreate(&stream1));
    cudaCheckError(hipStreamCreate(&stream2));

    // Allocate device memory
    cudaCheckError(hipMalloc(&d_grid_current, columnLength * rowLength * sizeof(int)));
    cudaCheckError(hipMalloc(&d_grid_next, columnLength * rowLength * sizeof(int)));

    // Asynchronously copy the initial state to the device using stream1
    cudaCheckError(hipMemcpyAsync(d_grid_current, hostGrid.data(), columnLength * rowLength * sizeof(int), hipMemcpyHostToDevice, stream1));

    hipEvent_t start, stop;
    cudaCheckError(hipEventCreate(&start));
    cudaCheckError(hipEventCreate(&stop));

    float elapsedTime = 0.0f;
    int iterationCount = 0;

    while (window.isOpen()) {
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window.close();
                break;
            }
        }

        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
            window.close();
            break;
        }

        // Record the start time of kernel execution
        cudaCheckError(hipDeviceSynchronize());
        cudaCheckError(hipEventRecord(start, stream1));

        // Launch the kernel using stream1
        updateSingleGridKernelWithHalo<<<numBlocks, numThreadPerBlock, 0, stream1>>>(d_grid_current, d_grid_next, columnLength, rowLength);

        // Asynchronously copy the result back to host memory using stream2
        cudaCheckError(hipMemcpyAsync(hostGrid.data(), d_grid_current, columnLength * rowLength * sizeof(int), hipMemcpyDeviceToHost, stream2));

        // Record the stop time and synchronize
        cudaCheckError(hipEventRecord(stop, stream1));
        cudaCheckError(hipEventSynchronize(stop));

        float iterationTime = 0.0f;
        cudaCheckError(hipEventElapsedTime(&iterationTime, start, stop));
        elapsedTime += iterationTime;

        // Print the elapsed time every 100 iterations
        if (++iterationCount % 100 == 0) {
            std::cout << "100 generations took " << elapsedTime << " ms with " << numThreadPerBlock << " threads per block using Normal memory allocation with streams\n";
            elapsedTime = 0.0f;
        }

        // Draw the grid using the data from hostGrid
        drawGrid(window, hostGrid.data());

        // Swap the grid buffers
        std::swap(d_grid_current, d_grid_next);
    }

    // Free device memory and destroy streams and events
    hipFree(d_grid_current);
    hipFree(d_grid_next);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Pinned memory transfer
void pinnedMemory(sf::RenderWindow& window) {
    hipStream_t stream1, stream2;
    cudaCheckError(hipStreamCreate(&stream1));
    cudaCheckError(hipStreamCreate(&stream2));

    // Allocate pinned host memory and device memory
    int* pinnedHostGrid;
    cudaCheckError(hipHostMalloc(&pinnedHostGrid, columnLength * rowLength * sizeof(int)));
    memcpy(pinnedHostGrid, hostGrid.data(), columnLength * rowLength * sizeof(int));

    cudaCheckError(hipMalloc(&d_grid_current, columnLength * rowLength * sizeof(int)));
    cudaCheckError(hipMalloc(&d_grid_next, columnLength * rowLength * sizeof(int)));

    // Asynchronously copy the initial state to the device using stream1
    cudaCheckError(hipMemcpyAsync(d_grid_current, pinnedHostGrid, columnLength * rowLength * sizeof(int), hipMemcpyHostToDevice, stream1));

    hipEvent_t start, stop;
    cudaCheckError(hipEventCreate(&start));
    cudaCheckError(hipEventCreate(&stop));

    float elapsedTime = 0.0f;
    int iterationCount = 0;

    while (window.isOpen()) {
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window.close();
                break;
            }
        }

        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
            window.close();
            break;
        }

        // Record the start time of kernel execution
        cudaCheckError(hipDeviceSynchronize());
        cudaCheckError(hipEventRecord(start, stream1));

        // Launch the kernel using stream1
        updateSingleGridKernelWithHalo<<<numBlocks, numThreadPerBlock, 0, stream1>>>(d_grid_current, d_grid_next, columnLength, rowLength);

        // Asynchronously copy the result back to pinned host memory using stream2
        cudaCheckError(hipMemcpyAsync(pinnedHostGrid, d_grid_current, columnLength * rowLength * sizeof(int), hipMemcpyDeviceToHost, stream2));
        
        // Record the stop time and synchronize
        cudaCheckError(hipEventRecord(stop, stream1));
        cudaCheckError(hipEventSynchronize(stop));

        float iterationTime = 0.0f;
        cudaCheckError(hipEventElapsedTime(&iterationTime, start, stop));
        elapsedTime += iterationTime;

        if (++iterationCount % 100 == 0) {
            std::cout << "100 generations took " << elapsedTime << " ms with " << numThreadPerBlock << " threads per block using Pinned memory allocation and streams\n";
            elapsedTime = 0.0f;
        }

        // Directly draw the latest grid using the pinned host memory
        drawGrid(window, pinnedHostGrid);

        // Swap the grid buffers
        std::swap(d_grid_current, d_grid_next);
    }

    // Free pinned host memory, device memory, destroy streams and events
    hipHostFree(pinnedHostGrid);
    hipFree(d_grid_current);
    hipFree(d_grid_next);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Managed memory transfer
void managedMemory(sf::RenderWindow& window) {
    // Allocate managed memory
    cudaCheckError(hipMallocManaged(&d_grid_current, columnLength * rowLength * sizeof(int)));
    cudaCheckError(hipMallocManaged(&d_grid_next, columnLength * rowLength * sizeof(int)));

    // Copy the initial state to managed memory
    cudaCheckError(hipMemcpy(d_grid_current, hostGrid.data(), columnLength * rowLength * sizeof(int), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    cudaCheckError(hipEventCreate(&start));
    cudaCheckError(hipEventCreate(&stop));

    float elapsedTime = 0.0f;
    int iterationCount = 0;

    while (window.isOpen()) {
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window.close();
                break;
            }
        }

        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
            window.close();
            break;
        }

        // Record the start time of kernel execution
        cudaCheckError(hipEventRecord(start, 0));

        // Launch the kernel using the default stream
        updateSingleGridKernelWithHalo<<<numBlocks, numThreadPerBlock>>>(d_grid_current, d_grid_next, columnLength, rowLength);

        // Synchronize to ensure kernel execution is complete
        cudaCheckError(hipDeviceSynchronize());

        // Record the stop time
        cudaCheckError(hipEventRecord(stop, 0));
        cudaCheckError(hipEventSynchronize(stop));

        float iterationTime = 0.0f;
        cudaCheckError(hipEventElapsedTime(&iterationTime, start, stop));
        elapsedTime += iterationTime;

        if (++iterationCount % 100 == 0) {
            std::cout << "100 generations took " << elapsedTime << " ms with " << numThreadPerBlock << " threads per block using Managed memory allocation\n";
            elapsedTime = 0.0f;
        }

        // Draw the grid using the updated host data
        drawGrid(window, d_grid_current);

        // Swap the grid buffers for the next iteration
        std::swap(d_grid_current, d_grid_next);
    }

    // Free managed memory and destroy events
    hipFree(d_grid_current);
    hipFree(d_grid_next);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char* argv[]) {
    // Obtain the basic arguments
    argumentsParse(argc, argv);

    // Create a window
    sf::RenderWindow window(sf::VideoMode(windowWeight, windowHeight), "John Conway\'s Game of Life");
    
    // Initialize grid size and number of blocks
    columnLength = windowWeight / cellSize;
    rowLength = windowHeight / cellSize;
    totalGrid = columnLength * rowLength;
    numBlocks = (totalGrid + numThreadPerBlock - 1) / numThreadPerBlock;

    // Use randon seed to initialize grid
    std::srand(RAND_SEED);
    hostGrid.reserve(totalGrid);
    for (int i = 0; i < totalGrid; i++) {
        hostGrid[i] = std::rand() % 2;
    }

    // Call the different types of memory transfer
    switch (memoryType) {
        case MemoryType::NORMAL:
            normalMemory(window);
            break;
        case MemoryType::PINNED:
            pinnedMemory(window);
            break;
        case MemoryType::MANAGED:
            managedMemory(window);
            break;
        default:
            std::cout << "Undefined Memory Type. Please choose one from NOMRAL, PINNED, MANAGED" << std::endl;
    }
    
    return 0;
}
