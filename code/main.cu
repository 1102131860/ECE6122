#include "hip/hip_runtime.h"
// main.cpp
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include "cuda_kernels.cuh"
#include <iostream>
#include <vector>
#include <string>
#include <ctime>
#include <chrono>
#include <cstdlib>

#define DEFAULT_NUM_THREAD_PER_BLOCK 32
#define DEFAULT_CELL_SIZE 5
#define DEFAUlT_WINDOW_WEIGHT 800
#define DEFAULT_WINDOW_HEIGHT 600
#define RAND_SEED 42
#define RADIUS 1

enum class MemoryType {
	NORMAL = 0,
	PINNED = 1,
	MANAGED = 2
};

int numThreadPerBlock, cellSize, windowWeight, windowHeight;
MemoryType memoryType;

int columnLength, rowLength, totalGrid;
std::vector<int> hostGrid;
int* d_grid_current, d_grid_next;

// Obtain arguments through inputs
void argumentsParse(int argc, char* argv[]) {
    for (int i = 1; i < argc; i += 2) {
        std::string argKey = argv[i];
        std::string argValue = (i + 1 < argc) ? argv[i + 1] : "";

        if (argKey == "-n" && !argValue.empty()) {
            try {
                numThreadPerBlock = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-n): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
                std::cout << "Out of range (-n): " << e.what() << std::endl;
            }
            if (numThreadPerBlock <= 0 || numThreadPerBlock % 32 != 0) {
                std::cout << "The number of threads per block must be a multiple of 32" << std::endl;
                numThreadPerBlock = DEFAULT_NUM_THREAD_PER_BLOCK;
            }
        } else if (argKey == "-c" && !argValue.empty()) {
            try {
	            cellSize = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-c): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
	            std::cout << "Out of range (-c): " << e.what() << std::endl;
            }
            if (cellSize < 1) {
                std::cout << "Cell size must be larger than or equal to 1" << std::endl;
                cellSize = DEFAULT_CELL_SIZE;
            }
        } else if (argKey == "-x" && !argValue.empty()) {
            try {
                windowWeight = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-x): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
                std::cout << "Out of range (-x): " << e.what() << std::endl;
            } if (windowWeight < cellSize) {
                std::cout << "Winodw weight mush be larger than or equal to cell size" << std::endl;
                windowWeight = DEFAUlT_WINDOW_WEIGHT;
            }
        } else if (argKey == "-y" && !argValue.empty()) {
            try {
                windowHeight = std::stoi(argValue);
            } catch (const std::invalid_argument& e) {
                std::cout << "Invalid argument (-y): " << e.what() << std::endl;
            } catch (const std::out_of_range& e) {
                std::cout << "Out of range (-y): " << e.what() << std::endl;
            }
            if (windowHeight < cellSize) {
                std::cout << "Winodw height mush be larger than or equal to cell size" << std::endl;
                windowHeight = DEFAULT_WINDOW_HEIGHT;
            }
        } else if (argKey == "-t" && !argValue.empty()) {
            if (argValue == "NORMAL"){
                memoryType = MemoryType::NORMAL;
            } else if (argValue == "THRD") {
                memoryType = MemoryType::PINNED;
            } else if (argValue == "MANAGED") {
                memoryType == MemoryType::MANAGED;
            } else {
                std::cout << "Invalid argument (-t): select a type from NORMAL, PINNED and MANAGED" << std::endl;
                memoryType = MemoryType::NORMAL;
            }
        }
    }
}

// Update 1-D Grid with Halo padding (share memory) 
__global__ void updateSingleGridKernelWithHalo(int* grid, int* newGrid, int columnLength, int rowLength) {
    // Use extern shared memory for dynamic allocation with halo padding
    extern __shared__ int sharedGrid[];

    // Calculate the dimensions of the block and the shared memory size including halo
    int sharedWidth = blockDim.x + 2 * RADIUS;

    // Calculate global and local indices
    int blockColumn = blockIdx.x * blockDim.x;
    int blockRow = blockIdx.y * blockDim.y;

    int localX = threadIdx.x;
    int localY = threadIdx.y;

    int globalX = blockColumn + localX;
    int globalY = blockRow + localY;

    int sharedX = localX + RADIUS;
    int sharedY = localY + RADIUS;

    // Load the main cell and halo cells into shared memory using a unified loop
    for (int dy = -RADIUS; dy <= RADIUS; dy++) {
        for (int dx = -RADIUS; dx <= RADIUS; dx++) {
            // Calculate the global and shared memory indices for each cell to be loaded
            int haloGlobalX = globalX + dx;
            int haloGlobalY = globalY + dy;
            int haloSharedX = sharedX + dx;
            int haloSharedY = sharedY + dy;

            // Only load the cells if the current thread is responsible for them
            if (localX + dx >= -RADIUS && localX + dx < blockDim.x + RADIUS &&
                localY + dy >= -RADIUS && localY + dy < blockDim.y + RADIUS) {

                // Check if within the bounds of the global grid and load appropriately
                if (haloGlobalX >= 0 && haloGlobalX < columnLength && haloGlobalY >= 0 && haloGlobalY < rowLength) {
                    sharedGrid[haloSharedY * sharedWidth + haloSharedX] = grid[haloGlobalY * columnLength + haloGlobalX];
                } else {
                    sharedGrid[haloSharedY * sharedWidth + haloSharedX] = 0; // Handle out-of-bounds by setting to 0
                }
            }
        }
    }

    // Synchronize to ensure all cells, including halo, are loaded into shared memory
    __syncthreads();

    // Calculate the number of live neighbors
    int numOfLiveNeighbours = 0;

    // Count neighbors
    for (int dy = -RADIUS; dy <= RADIUS; dy++) {
        for (int dx = -RADIUS; dx <= RADIUS; dx++) {
            if (dx == 0 && dy == 0) {
                continue; // Skip the center cell itself
            }
            numOfLiveNeighbours += sharedGrid[(sharedY + dy) * sharedWidth + (sharedX + dx)];
        }
    }

    // Apply the rules of the game and write to the new grid
    if (sharedGrid[sharedY * sharedWidth + sharedX] == 1) {
        newGrid[globalY * columnLength + globalX] = (numOfLiveNeighbours == 2 || numOfLiveNeighbours == 3) ? 1 : 0;
    } else {
        newGrid[globalY * columnLength + globalX] = (numOfLiveNeighbours == 3) ? 1 : 0;
    }
}

// Draw the gird using RectangleShape
void drawGrid(sf::RenderWindow window) {
    window.clear(sf::Color::Black);
    for (int x = 0; x < columnLength; ++x) {
        for (int y = 0; y < rowLength; ++y) {
            if (hostGrid[y * columnLength + x] == 1) {
                sf::RectangleShape cell(sf::Vector2f(cellSize, cellSize));
                cell.setPosition(x * cellSize, y * cellSize);
                cell.setFillColor(sf::Color::White);
                window.draw(cell);
            }
        }
    }
    window.display();
}

void runIterations() {
    // Create a window
    sf::RenderWindow window(sf::VideoMode(windowWeight, windowHeight), "John Conway\'s Game of Life");

    // Allocate device memory
    hipMalloc(&d_grid_current, columnLength * rowLength * sizeof(int));
    hipMalloc(&d_grid_next, columnLength * rowLength * sizeof(int));

    // Copy the initial state to the device
    hipMemcpy(d_grid_current, hostGrid.data(), columnLength * rowLength * sizeof(int), hipMemcpyHostToDevice);

    // Define the block and grid size
    int blockSizeX = 32;  // Example block size
    int blockSizeY = numThreadPerBlock / blockSizeX;
    dim3 blockSize(blockSizeX, blockSizeY);
    dim3 gridSize((columnLength + blockSizeX - 1) / blockSizeX, (rowLength + blockSizeY - 1) / blockSizeY);
    size_t sharedMemSize = (blockSizeX + 2 * RADIUS) * (blockSizeY + 2 * RADIUS) * sizeof(int);

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsedTime = 0.0f;  // Accumulator for kernel execution time
    int iterationCount = 0;

    // Main loop to update the grid while the window is open
    while (window.isOpen()) {
        // Handle window events
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window.close();
                return;
            }
        }

        // If the Esc key is pressed, close the window
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
            window.close();
            return;
        }

        // Start timing for kernel execution
        hipEventRecord(start, 0);

        // Launch the kernel
        updateSingleGridKernelWithHalo<<<gridSize, blockSize, sharedMemSize>>>(d_grid_current, d_grid_next, columnLength, rowLength);
        hipDeviceSynchronize();

        // Stop timing for kernel execution
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float iterationTime = 0.0f;
        hipEventElapsedTime(&iterationTime, start, stop);
        elapsedTime += iterationTime;

        // Swap the buffers
        std::swap(d_grid_current, d_grid_next);

        // Print elapsed time for every 100 iterations
        if (++iterationCount % 100 == 0) {
            std::cout << "Time for 100 iterations: " << elapsedTime << " ms" << std::endl;
            elapsedTime = 0.0f; // Reset elapsed time for the next 100 iterations
        }

        // Copy the current grid to host memory for visualization
        hipMemcpy(hostGrid.data(), d_grid_current, columnLength * rowLength * sizeof(int), hipMemcpyDeviceToHost);

        // Draw the grid using RectangleShape
        drawGrid(window);
    }

    // Free device memory
    hipFree(d_grid_current);
    hipFree(d_grid_next);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int agrc, char* argv[]) {
    // Obtain the basic arguments
    argumentsParse(argc, agrv);
    std::cout << "numThreadPerBlock: " << numThreadPerBlock << " cellSize: " << cellSize << " windowWeight: "
	    << windowWeight << " windowHeight: " << windowHeight << " MemoryType: "
	    << (MemoryType == MemoryType::NORMAL ? "NORMAL" : (executionType == MemoryType::PINNED ? "PINNED" : "MANAGED"))
	    << std::endl;
    
    // Create a window
    sf::RenderWindow window(sf::VideoMode(windowWeight, windowHeight), "John Conway\'s Game of Life");

    // Initilize grid
    columnLength = windowWeight / cellSize;
    rowLength = windowHeight / cellSize;
    totalGrid = columnLength * rowLength;

    // Use randon seed to initial grid
    std::srand(RAND_SEED);
    hostGrid.reserve(totalGrid);
    for (int i = 0; i < totalGrid; i++) {
        hostGrid[i] = std::rand() % 2;
    }
    
    return 0;
}
